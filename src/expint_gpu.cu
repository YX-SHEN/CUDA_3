#include "expint_gpu.hpp"
#include <hip/hip_runtime.h>
#include <cstdio>

namespace gpu {

// 申请 device 内存并 H2D 拷贝
void alloc_and_copy_to_device(const float* h_x, float*& d_x, int samples) {
    size_t bytes = samples * sizeof(float);
    hipError_t err;
    err = hipMalloc((void**)&d_x, bytes);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(err));
        d_x = nullptr;
        return;
    }
    err = hipMemcpy(d_x, h_x, bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMemcpy (H2D) failed: %s\n", hipGetErrorString(err));
        hipFree(d_x);
        d_x = nullptr;
    }
}

void free_device(float* d_x) {
    if (d_x) hipFree(d_x);
}

} // namespace gpu
